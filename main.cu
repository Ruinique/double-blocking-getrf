#include "hip/hip_runtime.h"
#include <fmt/core.h>

#include <stdexcept>  // For std::runtime_error
#include <utility>

#include "external/Qui1Framework/include/common/error_check.cuh"
#include "external/Qui1Framework/include/matrix/qui1_device_matrix.cuh"
#include "external/Qui1Framework/include/matrix/qui1_matrix_helper.cuh"
#include "external/Qui1Framework/include/wrapper/solver/qui1_cusolver_wrapper.cuh"
#include "external/Qui1Framework/include/common/device_info_print.cuh"

using data_type = float;

int main(int argc, char** argv) {
    qui1::common::print_device_info();
    {
        const size_t N = 32;
        fmt::print("Matrix size N = {}\n", N);
        // 1. Create matrix and fill with random data
        qui1::DeviceMatrix<data_type> A(N, N);
        qui1::MatrixHelper::fillWithRandom(A);
        // 2. Get a view (using right-value reference)
        auto&& A_view = A.getView(4, 4, 1, 1);
        qui1::MatrixHelper::printMatrix(A_view);
        // 3. Prepare for LU decomposition
        qui1::CusolverWrapper solver;
        int* devIpiv = nullptr;
        const int min_mn = static_cast<int>(std::min(A.getRows(), A.getCols()));
        CUDA_CHECK(
            hipMalloc(reinterpret_cast<void**>(&devIpiv), sizeof(int) * min_mn));
        // 4. Perform LU decomposition
        solver.getrf(A_view, devIpiv);
        qui1::MatrixHelper::printMatrix(A_view);
        // 5. Clean up
        if (devIpiv) {
            CUDA_CHECK(hipFree(devIpiv));
        }
    }
    {

    }
    return 0;
}
